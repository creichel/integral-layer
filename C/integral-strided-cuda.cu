#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <algorithm>

#include <hip/hip_runtime.h>

#include <THC/THC.h>

#define BLOCK_SIZE 32
#define BLOCK_CHANNELS (1024 / (BLOCK_SIZE * BLOCK_SIZE))

using std::max;
using std::min;
using std::floor;
using std::ceil;

// TODO remove this code
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

namespace strided {

/************************ updateOutput ************************/

// Divides x by y (y > 0), rounds towards minus infinity 
__device__ inline
int divFloor(const int x, const int y) {
    return x >= 0 ? x / y : (x - y + 1) / y;
}

// Divides x by y (y > 0), rounds towards minus infinity, returns positive remainder
__device__ inline
int modFloor(const int x, const int y) {
    return x >= 0 ? x % y : (y + x % y);
}

__global__ void forwardNoNormReplicateKernel(
    const float *intData, const int intDataStrideChannel, float *outData,
    const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w,
    const float *const xMin, const float *const xMax,
    const float *const yMin, const float *const yMax,
    const int strideH, const int strideW) {

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    outData += id; // outData now points to our output pixel

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    const int y = id % wOut; id /= wOut;
    const int x = id % hOut; id /= hOut;
    const int windowIdx = id % nWindows; id /= nWindows;

    // `id` is now is now the current global input plane number
    intData += id * intDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;

    if (batchIdx < batchSize) {

        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at points 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.

        const int t = max(0, min(x*strideH+(int) ceil(xMin[globalWindowIdx])  , h-1) );
        const int b = max(1, min(x*strideH+(int)floor(xMax[globalWindowIdx])+1, h  ) );
        const int l = max(0, min(y*strideW+(int) ceil(yMin[globalWindowIdx])  , w-1) );
        const int r = max(1, min(y*strideW+(int)floor(yMax[globalWindowIdx])+1, w  ) );

        float outValue = 0;

        outValue += intData[b*(w+1) + r];
        outValue -= intData[t*(w+1) + r];
        outValue -= intData[b*(w+1) + l];
        outValue += intData[t*(w+1) + l];

        *outData = outValue;
    }
}

__global__ void forwardNoNormReplicateFracKernel(
    const float *intData, const int intDataStrideChannel, float *outData,
    const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w,
    const float *const xMin, const float *const xMax,
    const float *const yMin, const float *const yMax,
    const float *inData, const int inDataStrideRow, const int inDataStrideChannel,
    const int strideH, const int strideW) {

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    outData += id; // outData now points to our output pixel

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    const int y = id % wOut; id /= wOut;
    const int x = id % hOut; id /= hOut;
    const int windowIdx = id % nWindows; id /= nWindows;

    // `id` is now is now the current global input plane number
    intData += id * intDataStrideChannel;
    inData  += id *  inDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;

    if (batchIdx < batchSize) {

        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at points 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.

        const int   xMinCurr = (int)ceil(xMin[globalWindowIdx]);
        const float xMinCurrFrac = (float)xMinCurr - xMin[globalWindowIdx];
        const int   yMinCurr = (int)ceil(yMin[globalWindowIdx]);
        const float yMinCurrFrac = (float)yMinCurr - yMin[globalWindowIdx];

        const float xMaxCurrFrac = xMax[globalWindowIdx] - floor(xMax[globalWindowIdx]);
        const int   xMaxCurr = (int)floor(xMax[globalWindowIdx]) + 1;
        const float yMaxCurrFrac = yMax[globalWindowIdx] - floor(yMax[globalWindowIdx]);
        const int   yMaxCurr = (int)floor(yMax[globalWindowIdx]) + 1;

        const int t = max(0, min(x*strideH+xMinCurr, h-1) );
        const int b = max(1, min(x*strideH+xMaxCurr, h)   );
        const int l = max(0, min(y*strideW+yMinCurr, w-1) );
        const int r = max(1, min(y*strideW+yMaxCurr, w)   );

        double outValue = 0;

        outValue += intData[b*(w+1) + r];
        outValue -= intData[t*(w+1) + r];
        outValue -= intData[b*(w+1) + l];
        outValue += intData[t*(w+1) + l];

        // TODO: tAdv, bAdv, lAdv, rAdv
        // -- xMax border
        outValue +=
            ( intData[max(1,min(x*strideH+xMaxCurr+1,h))*(w+1) 
                + max(1,min(y*strideW+yMaxCurr,w))]
            - intData[max(1,min(x*strideH+xMaxCurr,h))*(w+1)
                + max(1,min(y*strideW+yMaxCurr,w))]
            - intData[max(1,min(x*strideH+xMaxCurr+1,h))*(w+1)
                + max(0,min(y*strideW+yMinCurr,w-1))]
            + intData[max(1,min(x*strideH+xMaxCurr,h))*(w+1)
                + max(0,min(y*strideW+yMinCurr,w-1))]
            ) * xMaxCurrFrac;

        // -- yMax border
        outValue +=
            ( intData[max(1,min(x*strideH+xMaxCurr,h))*(w+1) 
                + max(1,min(y*strideW+yMaxCurr+1,w))]
            - intData[max(1,min(x*strideH+xMaxCurr,h))*(w+1)
                + max(1,min(y*strideW+yMaxCurr,w))]
            - intData[max(0,min(x*strideH+xMinCurr,h-1))*(w+1)
                + max(1,min(y*strideW+yMaxCurr+1,w))]
            + intData[max(0,min(x*strideH+xMinCurr,h-1))*(w+1)
                + max(1,min(y*strideW+yMaxCurr,w))]
            ) * yMaxCurrFrac;

        // -- xMin border
        outValue +=
            ( intData[max(0,min(x*strideH+xMinCurr,h-1))*(w+1) 
                + max(1,min(y*strideW+yMaxCurr,w))]
            - intData[max(0,min(x*strideH+xMinCurr-1,h-1))*(w+1)
                + max(1,min(y*strideW+yMaxCurr,w))]
            - intData[max(0,min(x*strideH+xMinCurr,h-1))*(w+1)
                + max(0,min(y*strideW+yMinCurr,w-1))]
            + intData[max(0,min(x*strideH+xMinCurr-1,h-1))*(w+1)
                + max(0,min(y*strideW+yMinCurr,w-1))]
            ) * xMinCurrFrac;

        // -- yMin border
        outValue +=
            ( intData[max(1,min(x*strideH+xMaxCurr,h))*(w+1) 
                + max(0,min(y*strideW+yMinCurr,w-1))]
            - intData[max(1,min(x*strideH+xMaxCurr,h))*(w+1)
                + max(0,min(y*strideW+yMinCurr-1,w-1))]
            - intData[max(0,min(x*strideH+xMinCurr,h-1))*(w+1)
                + max(0,min(y*strideW+yMinCurr,w-1))]
            + intData[max(0,min(x*strideH+xMinCurr,h-1))*(w+1)
                + max(0,min(y*strideW+yMinCurr-1,w-1))]
            ) * yMinCurrFrac;

        // -- corner pixels
        outValue += 
            xMaxCurrFrac*yMaxCurrFrac * (
               (x*strideH+xMaxCurr >  h-1 or
                y*strideW+yMaxCurr >  w-1 or
                x*strideH+xMaxCurr <= 0   or
                y*strideW+yMaxCurr <= 0) ? 0 : 
                    inData[(x*strideH+xMaxCurr)*inDataStrideRow + (y*strideW+yMaxCurr)]);

        outValue +=
            xMinCurrFrac*yMaxCurrFrac * (
               (x*strideH+xMinCurr-1 >= h-1 or
                y*strideW+yMaxCurr   >  w-1 or
                x*strideH+xMinCurr-1 <  0   or
                y*strideW+yMaxCurr   <= 0) ? 0 : 
                    inData[(x*strideH+xMinCurr-1)*inDataStrideRow + (y*strideW+yMaxCurr)]);

        outValue +=
            xMaxCurrFrac*yMinCurrFrac * (
               (x*strideH+xMaxCurr   >  h-1 or
                y*strideW+yMinCurr-1 >= w-1 or
                x*strideH+xMaxCurr   <= 0   or
                y*strideW+yMinCurr-1 <  0) ? 0 : 
                    inData[(x*strideH+xMaxCurr)*inDataStrideRow + (y*strideW+yMinCurr-1)]);

        outValue +=
            xMinCurrFrac*yMinCurrFrac * (
               (x*strideH+xMinCurr-1 >= h-1 or
                y*strideW+yMinCurr-1 >= w-1 or
                x*strideH+xMinCurr-1 <  0   or
                y*strideW+yMinCurr-1 <  0) ? 0 : 
                    inData[(x*strideH+xMinCurr-1)*inDataStrideRow + (y*strideW+yMinCurr-1)]);
        
        *outData = outValue;
    }
}

void forwardNoNormReplicateCuda(THCState *state,
    const float *intData, const int intDataStrideChannel, float *outData,
    const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMin, const float *yMax,
    const int strideH, const int strideW) {

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    const int NUM_THREADS = BLOCK_SIZE * BLOCK_SIZE;
    const int threadsNeeded = batchSize * nInputPlane * nWindows * hOut * wOut;
    const int numBlocks = (threadsNeeded + NUM_THREADS - 1) / NUM_THREADS;
    
    forwardNoNormReplicateKernel
        <<<numBlocks, NUM_THREADS, 0, THCState_getCurrentStream(state)>>> (
        intData, intDataStrideChannel, outData,
        batchSize, nInputPlane, nWindows, h, w,
        xMin, xMax, yMin, yMax,
        strideH, strideW);
    THCudaCheck(hipGetLastError());
}

void forwardNoNormReplicateFracCuda(THCState *state,
    const float *intData, const int intDataStrideChannel, float *outData,
    const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMin, const float *yMax,
    const float *inData, const int inDataStrideRow, const int inDataStrideChannel,
    const int strideH, const int strideW) {

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    const int NUM_THREADS = BLOCK_SIZE * BLOCK_SIZE;
    const int threadsNeeded = batchSize * nInputPlane * nWindows * hOut * wOut;
    const int numBlocks = (threadsNeeded + NUM_THREADS - 1) / NUM_THREADS;

    forwardNoNormReplicateFracKernel
        <<<numBlocks, NUM_THREADS, 0, THCState_getCurrentStream(state)>>> (
        intData, intDataStrideChannel, outData,
        batchSize, nInputPlane, nWindows, h, w,
        xMin, xMax, yMin, yMax,
        inData, inDataStrideRow, inDataStrideChannel,
        strideH, strideW);
    THCudaCheck(hipGetLastError());
}

/************************ updateGradInput ************************/

__global__ void updateGradInputReplicatePlanewiseKernel(
    const float *gradOutputIntData, float * const gradInputData,
    const int h, const int w, const int nWindows,
    const float * const xMin, const float * const xMax,
    const float * const yMin, const float * const yMax,
    const int strideH, const int strideW) {

    const int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    if (x < h and y < w) {

        int xMinCurr, xMaxCurr, yMinCurr, yMaxCurr;
        double outValue = 0;

        for (int windowIdx = 0; windowIdx < nWindows; ++windowIdx) {

            xMinCurr = (int)ceil(-xMax[windowIdx]);
            yMinCurr = (int)ceil(-yMax[windowIdx]);

            xMaxCurr = (int)floor(-xMin[windowIdx]) + 1;
            yMaxCurr = (int)floor(-yMin[windowIdx]) + 1;

            // The following code block implements these lines
            // as if they were executed simultaneously (see `void updateGradInputFrac()`):
            // xMinCurr = (x == 0   and xMaxCurr >= 0 ? 0    : xMinCurr);
            // xMaxCurr = (x == h-1 and xMinCurr <= 0 ? h+66 : xMaxCurr);
            // yMinCurr = (y == 0   and yMaxCurr >= 0 ? 0    : yMinCurr);
            // yMaxCurr = (y == w-1 and yMinCurr <= 0 ? w+66 : yMaxCurr);

            bool needToChangeMin, needToChangeMax;

            needToChangeMin = x == 0   and xMaxCurr >= 0;
            needToChangeMax = x == h-1 and xMinCurr <= 0;
            if (needToChangeMin) xMinCurr = 0;
            if (needToChangeMax) xMaxCurr = h+66;

            needToChangeMin = y == 0   and yMaxCurr >= 0;
            needToChangeMax = y == w-1 and yMinCurr <= 0;
            if (needToChangeMin) yMinCurr = 0;
            if (needToChangeMax) yMaxCurr = w+66;

            const int t = max(0, min(divFloor(x+xMinCurr + strideH - 1, strideH)    , hOut) );
            const int b = max(0, min(divFloor(x+xMaxCurr - 1          , strideH) + 1, hOut) );
            const int l = max(0, min(divFloor(y+yMinCurr + strideW - 1, strideW)    , wOut) );
            const int r = max(0, min(divFloor(y+yMaxCurr - 1          , strideW) + 1, wOut) );

            outValue += gradOutputIntData[b*(wOut+1) + r];
            outValue -= gradOutputIntData[t*(wOut+1) + r];
            outValue -= gradOutputIntData[b*(wOut+1) + l];
            outValue += gradOutputIntData[t*(wOut+1) + l];

            // go to the next channel
            gradOutputIntData += (hOut+1)*(wOut+1);
        }

        gradInputData[x*w + y] = outValue;
    }
}

__global__ void updateGradInputReplicatePlanewiseFracKernel(
    const float *gradOutputIntData, float * const gradInputData,
    const int h, const int w, const int nWindows,
    const float * const xMin, const float * const xMax,
    const float * const yMin, const float * const yMax,
    const float *gradOutputData, const int gradOutputStrideRow,
    const int gradOutputStrideChannel,
    const int strideH, const int strideW) {

    const int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    if (x < h and y < w) {

        int xMinCurr, xMaxCurr, yMinCurr, yMaxCurr;
        double outValue = 0;

        for (int windowIdx = 0; windowIdx < nWindows; ++windowIdx) {

            xMinCurr = (int)ceil(-xMax[windowIdx]);
            yMinCurr = (int)ceil(-yMax[windowIdx]);
            const float xMinCurrFrac = (float)xMinCurr + xMax[windowIdx];
            const float yMinCurrFrac = (float)yMinCurr + yMax[windowIdx];

            xMaxCurr = (int)floor(-xMin[windowIdx]) + 1;
            yMaxCurr = (int)floor(-yMin[windowIdx]) + 1;
            const float xMaxCurrFrac = -xMin[windowIdx] + 1 - xMaxCurr;
            const float yMaxCurrFrac = -yMin[windowIdx] + 1 - yMaxCurr;

            // The following code block implements these lines
            // as if they were executed simultaneously (see `void updateGradInputFrac()`):
            // xMinCurr = (x == 0   and xMaxCurr >= 0 ? 0    : xMinCurr);
            // xMaxCurr = (x == h-1 and xMinCurr <= 0 ? h+66 : xMaxCurr);
            // yMinCurr = (y == 0   and yMaxCurr >= 0 ? 0    : yMinCurr);
            // yMaxCurr = (y == w-1 and yMinCurr <= 0 ? w+66 : yMaxCurr);

            bool needToChangeMin, needToChangeMax;

            needToChangeMin = x == 0   and xMaxCurr >= 0;
            needToChangeMax = x == h-1 and xMinCurr <= 0;
            if (needToChangeMin) xMinCurr = 0;
            if (needToChangeMax) xMaxCurr = h+66;

            needToChangeMin = y == 0   and yMaxCurr >= 0;
            needToChangeMax = y == w-1 and yMinCurr <= 0;
            if (needToChangeMin) yMinCurr = 0;
            if (needToChangeMax) yMaxCurr = w+66;

            const int t = max(0, min(divFloor(x+xMinCurr + strideH - 1, strideH)    , hOut) );
            const int b = max(0, min(divFloor(x+xMaxCurr - 1          , strideH) + 1, hOut) );
            const int l = max(0, min(divFloor(y+yMinCurr + strideW - 1, strideW)    , wOut) );
            const int r = max(0, min(divFloor(y+yMaxCurr - 1          , strideW) + 1, wOut) );

            const int tAdv = modFloor(x+xMinCurr-1, strideH) == 0 and x+xMinCurr-1 <  h ? max(0, min(t-1, hOut)) : t;
            const int bAdv = modFloor(x+xMaxCurr  , strideH) == 0 and x+xMaxCurr   >= 0 ? max(0, min(b+1, hOut)) : b;
            const int lAdv = modFloor(y+yMinCurr-1, strideW) == 0 and y+yMinCurr-1 <  w ? max(0, min(l-1, wOut)) : l;
            const int rAdv = modFloor(y+yMaxCurr  , strideW) == 0 and y+yMaxCurr   >= 0 ? max(0, min(r+1, wOut)) : r;

            // TODO: 1D grid
            outValue += gradOutputIntData[b*(wOut+1) + r];
            outValue -= gradOutputIntData[t*(wOut+1) + r];
            outValue -= gradOutputIntData[b*(wOut+1) + l];
            outValue += gradOutputIntData[t*(wOut+1) + l];

            // -- xMax border
            outValue +=
                ( gradOutputIntData[bAdv*(wOut+1) + r]
                - gradOutputIntData[b   *(wOut+1) + r]
                - gradOutputIntData[bAdv*(wOut+1) + l]
                + gradOutputIntData[b   *(wOut+1) + l]
                ) * xMaxCurrFrac;

            // -- yMax border
            outValue +=
                ( gradOutputIntData[b*(wOut+1) + rAdv]
                - gradOutputIntData[b*(wOut+1) + r   ]
                - gradOutputIntData[t*(wOut+1) + rAdv]
                + gradOutputIntData[t*(wOut+1) + r   ]
                ) * yMaxCurrFrac;

            // -- xMin border
            outValue +=
                ( gradOutputIntData[t   *(wOut+1) + r]
                - gradOutputIntData[tAdv*(wOut+1) + r]
                - gradOutputIntData[t   *(wOut+1) + l]
                + gradOutputIntData[tAdv*(wOut+1) + l]
                ) * xMinCurrFrac;

            // -- yMin border
            outValue +=
                ( gradOutputIntData[b*(wOut+1) + l   ]
                - gradOutputIntData[b*(wOut+1) + lAdv]
                - gradOutputIntData[t*(wOut+1) + l   ]
                + gradOutputIntData[t*(wOut+1) + lAdv]
                ) * yMinCurrFrac;

            // -- corner pixels
            outValue += 
                xMaxCurrFrac*yMaxCurrFrac * (
                   (x+xMaxCurr > h-1 or
                    y+yMaxCurr > w-1 or
                    x+xMaxCurr < 0   or
                    y+yMaxCurr < 0   or
                    b == bAdv or
                    r == rAdv) ? 0 : 
                    gradOutputData[b*gradOutputStrideRow + r]);

            outValue +=
                xMinCurrFrac*yMaxCurrFrac * (
                   (x+xMinCurr-1 > h-1 or
                    y+yMaxCurr   > w-1 or
                    x+xMinCurr-1 < 0   or
                    y+yMaxCurr   < 0   or
                    t == tAdv or
                    r == rAdv) ? 0 : 
                    gradOutputData[tAdv*gradOutputStrideRow + r]);

            outValue +=
                xMaxCurrFrac*yMinCurrFrac * (
                   (x+xMaxCurr   > h-1 or
                    y+yMinCurr-1 > w-1 or
                    x+xMaxCurr   < 0   or
                    y+yMinCurr-1 < 0   or
                    b == bAdv or
                    l == lAdv) ? 0 : 
                    gradOutputData[b*gradOutputStrideRow + lAdv]);

            outValue +=
                xMinCurrFrac*yMinCurrFrac * (
                   (x+xMinCurr-1 > h-1 or
                    y+yMinCurr-1 > w-1 or
                    x+xMinCurr-1 < 0   or
                    y+yMinCurr-1 < 0   or
                    t == tAdv or
                    l == lAdv) ? 0 : 
                    gradOutputData[tAdv*gradOutputStrideRow + lAdv]);

            // go to the next channel
            gradOutputIntData += (hOut+1)*(wOut+1);
            gradOutputData += gradOutputStrideChannel;
        }

        gradInputData[x*w + y] = outValue;
    }
}

void updateGradInputReplicatePlanewiseCuda(
    const float *gradOutputIntData, float * const gradInputData,
    const int h, const int w, const int nWindows,
    const float * const xMin, const float * const xMax,
    const float * const yMin, const float * const yMax,
    const int strideH, const int strideW) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_CHANNELS);
    dim3 dimGrid(
        (h + dimBlock.x - 1) / dimBlock.x, 
        (w + dimBlock.y - 1) / dimBlock.y);

    updateGradInputReplicatePlanewiseKernel <<<dimGrid, dimBlock>>> (
        gradOutputIntData, gradInputData,
        h, w, nWindows,
        xMin, xMax, yMin, yMax,
        strideH, strideW);
}

void updateGradInputReplicatePlanewiseFracCuda(
    const float *gradOutputIntData, float * const gradInputData,
    const int h, const int w, const int nWindows,
    const float *xMin, const float *xMax, const float *yMin, float *yMax,
    const float *gradOutputData, const int gradOutputStrideRow,
    const int gradOutputStrideChannel,
    const int strideH, const int strideW) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_CHANNELS);
    dim3 dimGrid(
        (h + dimBlock.x - 1) / dimBlock.x, 
        (w + dimBlock.y - 1) / dimBlock.y);

    updateGradInputReplicatePlanewiseFracKernel <<<dimGrid, dimBlock>>> (
        gradOutputIntData, gradInputData,
        h, w, nWindows,
        xMin, xMax, yMin, yMax,
        gradOutputData, gradOutputStrideRow, gradOutputStrideChannel,
        strideH, strideW);
}

/************************ accGradParameters ************************/

__global__ void xMaxDeltaIntegralReplicateFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMax, const float *yMin, const float *yMax,
    const float *inData, const int inDataStrideRow,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;
    
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        // const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        // const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        const int xMaxInt = (int)floor(xMax[windowIdx]);
        // const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        const int yMaxInt = (int)floor(yMax[windowIdx]);
        const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        // const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        // const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];
        const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += brCorner * (y+yMaxInt <  1 ? 1.0f : yMaxFrac);
        delta += blCorner * (y+yMinInt >= w ? 1.0f : yMinFrac);

        delta += 
            intData[max(0,min(x+xMaxInt+1, h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMaxInt  , h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMaxInt+1, h))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMaxInt  , h))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMaxInt >= 1 and x+xMaxInt < h);
        tmpArray[xOut*wOut + yOut] = delta;
    }
}

__global__ void xMinDeltaIntegralReplicateFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *yMin, const float *yMax,
    const float *inData, const int inDataStrideRow,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;
    
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        // const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        // const int xMaxInt = (int)floor(xMax[windowIdx]);
        // const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        const int yMaxInt = (int)floor(yMax[windowIdx]);
        const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        // const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];
        // const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += trCorner * (y+yMaxInt <  1 ? 1.0f : yMaxFrac);
        delta += tlCorner * (y+yMinInt >= w ? 1.0f : yMinFrac);

        delta += 
            intData[max(0,min(x+xMinInt  , h))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt-1, h))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt  , h))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMinInt-1, h))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMinInt >= 1 and x+xMinInt < h);
        tmpArray[xOut*wOut + yOut] *= -delta;
    }
}

__global__ void yMaxDeltaIntegralReplicateFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMax,
    const float *inData, const int inDataStrideRow,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;
    
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        // const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        // const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        const int yMaxInt = (int)floor(yMax[windowIdx]);
        // const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        // const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        // const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];
        const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += trCorner * (x+xMinInt >= h ? 1.0f : xMinFrac);
        delta += brCorner * (x+xMaxInt <  1 ? 1.0f : xMaxFrac);

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMaxInt+1, w))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMaxInt+1, w))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];

        delta *= (y+yMaxInt >= 1 and y+yMaxInt < w);
        tmpArray[xOut*wOut + yOut] = delta;
    }
}

__global__ void yMinDeltaIntegralReplicateFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMin,
    const float *inData, const int inDataStrideRow,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;
    
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        // const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        // const int yMaxInt = (int)floor(yMax[windowIdx]);
        // const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        // const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];
        // const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += tlCorner * (x+xMinInt >= h ? 1.0f : xMinFrac);
        delta += blCorner * (x+xMaxInt <  1 ? 1.0f : xMaxFrac);

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w))];

        delta *= (y+yMinInt >= 1 and y+yMinInt < w);
        tmpArray[xOut*wOut + yOut] *= -delta;
    }
}

void backwardReplicateFracCuda(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float * const xMin, const float * const xMax,
    const float * const yMin, const float * const yMax,
    const float *inData, const int inDataStrideRow,
    const int strideH, const int strideW) {

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid((nWindows * hOut * wOut + dimBlock.x - 1) / dimBlock.x);

    xMaxDeltaIntegralReplicateFracKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 0*nWindows*hOut*wOut, nWindows, h, w,
        xMax, yMin, yMax, inData, inDataStrideRow, strideH, strideW);

    xMinDeltaIntegralReplicateFracKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 1*nWindows*hOut*wOut, nWindows, h, w,
        xMin, yMin, yMax, inData, inDataStrideRow, strideH, strideW);

    yMaxDeltaIntegralReplicateFracKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 2*nWindows*hOut*wOut, nWindows, h, w,
        xMin, xMax, yMax, inData, inDataStrideRow, strideH, strideW);

    yMinDeltaIntegralReplicateFracKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 3*nWindows*hOut*wOut, nWindows, h, w,
        xMin, xMax, yMin, inData, inDataStrideRow, strideH, strideW);
}

__global__ void xMaxDeltaIntegralReplicateKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMax, const float *yMin, const float *yMax,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        // const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(1,min(x+xMaxInt+1, h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMaxInt  , h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(1,min(x+xMaxInt+1, h))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMaxInt  , h))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMaxInt >= 1 and x+xMaxInt < h);
        tmpArray[xOut*wOut + yOut] = delta;
    }
}

__global__ void xMinDeltaIntegralReplicateKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *yMin, const float *yMax,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;
    
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        // const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(0,min(x+xMinInt  , h-1))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt-1, h  ))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt  , h-1))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMinInt-1, h  ))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMinInt >= 1 and x+xMinInt < h);
        tmpArray[xOut*wOut + yOut] *= -delta;
    }
}

__global__ void yMaxDeltaIntegralReplicateKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMax,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;
    
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        // const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(1,min(y+yMaxInt+1, w))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(1,min(y+yMaxInt+1, w))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];

        delta *= (y+yMaxInt >= 1 and y+yMaxInt < w);
        tmpArray[xOut*wOut + yOut] = delta;
    }
}

__global__ void yMinDeltaIntegralReplicateKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMin,
    const int strideH, const int strideW) {
 
    // TODO: use block dim instead
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;
    
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

        const int x = xOut*strideH + 1;
        const int y = yOut*strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        // const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w  ))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w-1))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w  ))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w-1))];

        delta *= (y+yMinInt >= 1 and y+yMinInt < w);
        tmpArray[xOut*wOut + yOut] *= -delta;
    }
}

void backwardReplicateCuda(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float * const xMin, const float * const xMax,
    const float * const yMin, const float * const yMax,
    const int strideH, const int strideW) {

    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid((nWindows * hOut * wOut + dimBlock.x - 1) / dimBlock.x);

    xMaxDeltaIntegralReplicateKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 0*nWindows*hOut*wOut,
        nWindows, h, w, xMax, yMin, yMax, strideH, strideW);

    xMinDeltaIntegralReplicateKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 1*nWindows*hOut*wOut,
        nWindows, h, w, xMin, yMin, yMax, strideH, strideW);

    yMaxDeltaIntegralReplicateKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 2*nWindows*hOut*wOut,
        nWindows, h, w, xMin, xMax, yMax, strideH, strideW);

    yMinDeltaIntegralReplicateKernel <<<dimGrid, dimBlock>>> (
        intData, tmpArray + 3*nWindows*hOut*wOut,
        nWindows, h, w, xMin, xMax, yMin, strideH, strideW);
}

} // namespace
