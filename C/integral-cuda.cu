#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <algorithm>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <THC/THC.h>

#include "integral-strided-cuda.hpp"

#define BLOCK_SIZE 32
#define BLOCK_CHANNELS (1024 / (BLOCK_SIZE * BLOCK_SIZE))

using std::max;
using std::min;
using std::floor;
using std::ceil;

// TODO remove this code
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

/************************ Integral image computation ************************/

__global__ void accumulateRowsKernel(
    float *input, float *output, int channels, int totalRows, int w);
__global__ void accumulateColsKernel(
    float *input, float *output, int channels, int h, int w);
__global__ void accumulateColsInplaceKernel(
    float *input, int channels, int h, int w);
__global__ void accumulateColsInplaceTransposedKernel(
    float *input, int channels, int h, int w);

extern "C"
void integralImageCuda(THCState *state,
    float *input, float *output, int channels, int h, int w, float *tmp) {

    int blockSize1D, gridSize1D;
    float ONE = 1.0, ZERO = 0.0;

    hipblasSetStream(THCState_getCurrentBlasHandle(state), THCState_getCurrentStream(state));

    int totalCols = channels * w;
    blockSize1D = BLOCK_SIZE * BLOCK_SIZE;
    gridSize1D = (totalCols + blockSize1D - 1) / blockSize1D;
    accumulateColsKernel <<<gridSize1D, blockSize1D, 0, THCState_getCurrentStream(state)>>> 
        (input, output, channels, h, w);
    THCudaCheck(hipGetLastError());

    THCublasCheck(hipblasSgeam(
        THCState_getCurrentBlasHandle(state),
        HIPBLAS_OP_T, HIPBLAS_OP_N, channels * (h+1), w+1,
        &ONE, output, w+1,
        &ZERO, tmp, channels * (h+1),
        tmp, channels * (h+1)));

    int totalRows = channels * h;
    blockSize1D = BLOCK_SIZE * BLOCK_SIZE;
    gridSize1D = (totalRows + blockSize1D - 1) / blockSize1D;
    accumulateColsInplaceTransposedKernel
        <<<gridSize1D, blockSize1D, 0, THCState_getCurrentStream(state)>>> (tmp, channels, h, w);
    THCudaCheck(hipGetLastError());

    THCublasCheck(hipblasSgeam(
        THCState_getCurrentBlasHandle(state),
        HIPBLAS_OP_T, HIPBLAS_OP_N, w+1, channels * (h+1),
        &ONE, tmp, channels * (h+1),
        &ZERO, output, w+1,
        output, w+1));
}

/*
extern "C"
void integralImageInplaceCuda(float *input, float *output, int channels, int h, int w) {
    int blockSize1D, gridSize1D;

    int totalCols = channels * w;
    blockSize1D = BLOCK_SIZE * BLOCK_SIZE;
    gridSize1D = (totalCols + blockSize1D - 1) / blockSize1D;
    accumulateColsKernel <<<gridSize1D, blockSize1D>>> (input, output, channels, h, w);

    inplace::transpose(true, output, channels * (h+1), w+1);

    int totalRows = channels * h;
    blockSize1D = BLOCK_SIZE * BLOCK_SIZE;
    gridSize1D = (totalRows + blockSize1D - 1) / blockSize1D;
    accumulateColsInplaceTransposedKernel <<<gridSize1D, blockSize1D>>> (output, channels, h, w);

    inplace::transpose(true, output, w+1, channels * (h+1));
}
*/

__global__ void accumulateRowsKernel(
    float *input, float *output, int channels, int h, int w) {
    // view multichannel image as a multiline single-channel image
    int globalRowIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

    if (globalRowIdx < channels * h) {
        float *outputRow = output + (globalRowIdx + globalRowIdx / h + 1) * (w+1) + 1;
        outputRow[-1] = 0;

        double sum = 0;
        for (int i = 0; i < w; ++i) {
            sum += input[globalRowIdx * w + i];
            outputRow[i] = static_cast<float>(sum);
        }

        // need to zero the (0,0) corner of the output separately >:(
        output[(globalRowIdx / h) * (w+1) * (h+1)] = 0;
    }
}

__global__ void accumulateColsKernel(float *input, float *output, int channels, int h, int w) {
    // global column index (of all `channels * w` columns in this image)
    int colIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

    if (colIdx < channels * w) {
        // jump to current channel
        input  += (colIdx / w) * h * w;
        output += (colIdx / w) * (h+1) * (w+1);
        colIdx %= w; // switch to local column index,
        ++colIdx;    // it's 1-indexed because first output column is always zero

        output[colIdx] = 0; // first element of every column is always zero
        double sum = 0;

        for (int i = 1; i <= h; ++i) {
            sum += static_cast<double>(input[(i-1) * w + colIdx - 1]);
            output[i * (w+1) + colIdx] = static_cast<float>(sum);
        }
    }
}

__global__ void accumulateColsInplaceTransposedKernel(float *input, int channels, int h, int w) {
    // in-place.
    // input is a `(w+1) x channels * (h+1)` array

    // global column index (of all `channels * w` columns in this image)
    int colIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

    if (colIdx < channels * h) {
        // need to zero the (0,0) corner of the output separately >:(
        input[(colIdx / h) * (h+1)] = 0;

        colIdx += colIdx / h + 1; // make `colIdx` the (h+1)-array indexer

        input[colIdx] = 0; // first element of every column is always zero

        double sum = 0;

        for (int i = 1; i <= w; ++i) {
            float *currentElement = &input[i * channels * (h+1) + colIdx];
            sum += static_cast<double>(*currentElement);
            *currentElement = static_cast<float>(sum);
        }
    }
}

__global__ void accumulateColsInplaceKernel(float *input, int channels, int h, int w) {
    // in-place.
    // input is already a `channels * (h+1) x (w+1)` array

    // global column index (of all `channels * w` columns in this image)
    int colIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

    if (colIdx < channels * w) {
        input += (colIdx / w) * (h+1) * (w+1); // jump to current channel
        colIdx %= w; // switch to local column index,
        ++colIdx;    // it's 1-indexed because first output column is always zero

        input[colIdx] = 0; // first element of every column is always zero
        double sum = 0;

        for (int i = 1; i <= h; ++i) {
            float *currentElement = &input[i * (w+1) + colIdx];
            sum += static_cast<double>(*currentElement);
            *currentElement = static_cast<float>(sum);
        }
    }
}

/************************ updateOutput ************************/

__global__ void forwardKernel(
    float *intData, float *outData, int h, int w, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax, float *areaCoeff) {

    int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;
    int z = BLOCK_CHANNELS * blockIdx.z + threadIdx.z;

    if (x < h and y < w and z < nWindows) {

        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at indices 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.

        int t = max(0, min(x+(int)ceil (xMin[z]  ), h) );
        int b = max(0, min(x+(int)floor(xMax[z]+1), h) );
        int l = max(0, min(y+(int)ceil (yMin[z]  ), w) );
        int r = max(0, min(y+(int)floor(yMax[z]+1), w) );

        outData[z*w*h + x*w + y] = areaCoeff[z] *
            ( intData[b*(w+1) + r]
            - intData[t*(w+1) + r]
            - intData[b*(w+1) + l]
            + intData[t*(w+1) + l]);
    }
}

__global__ void forwardNoNormReplicateKernel(
    float *intData, int intDataStrideChannel, float *outData,
    int h, int w, int nInputPlane, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax) {

    const int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;
    const int z = BLOCK_CHANNELS * blockIdx.z + threadIdx.z;

    const int inPlaneIdx = z / nWindows;

    intData += intDataStrideChannel * inPlaneIdx;

    if (x < h and y < w and z < nInputPlane*nWindows) {

        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at points 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.

        const int t = max(0, min(x+(int) ceil(xMin[z])  , h-1) );
        const int b = max(1, min(x+(int)floor(xMax[z])+1, h  ) );
        const int l = max(0, min(y+(int) ceil(yMin[z])  , w-1) );
        const int r = max(1, min(y+(int)floor(yMax[z])+1, w  ) );

        double outValue = 0;

        outValue += intData[b*(w+1) + r];
        outValue -= intData[t*(w+1) + r];
        outValue -= intData[b*(w+1) + l];
        outValue += intData[t*(w+1) + l];

        outData[z*w*h + x*w + y] = outValue;
    }
}

__global__ void forwardNoNormReplicateFracKernel(
    float *intData, int intDataStrideChannel, float *outData,
    int h, int w, int nInputPlane, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax,
    float *inData, int inDataStrideRow, int inDataStrideChannel) {

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w; id /= w;
    const int x = id % h; id /= h;
    const int & z = id;

    const int inPlaneIdx = z / nWindows;

    intData += intDataStrideChannel * inPlaneIdx;
    inData  +=  inDataStrideChannel * inPlaneIdx;

    if (x < h and y < w and z < nInputPlane*nWindows) {

        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at points 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.

        const int   xMinCurr = (int)ceil(xMin[z]);
        const float xMinCurrFrac = (float)xMinCurr - xMin[z];
        const int   yMinCurr = (int)ceil(yMin[z]);
        const float yMinCurrFrac = (float)yMinCurr - yMin[z];

        const float xMaxCurrFrac = xMax[z] - floor(xMax[z]);
        const int   xMaxCurr = (int)floor(xMax[z]) + 1;
        const float yMaxCurrFrac = yMax[z] - floor(yMax[z]);
        const int   yMaxCurr = (int)floor(yMax[z]) + 1;

        const int t = max(0, min(x+xMinCurr, h-1) );
        const int b = max(1, min(x+xMaxCurr, h)   );
        const int l = max(0, min(y+yMinCurr, w-1) );
        const int r = max(1, min(y+yMaxCurr, w)   );

        double outValue = 0;

        outValue += intData[b*(w+1) + r];
        outValue -= intData[t*(w+1) + r];
        outValue -= intData[b*(w+1) + l];
        outValue += intData[t*(w+1) + l];

        // -- xMax border
        outValue +=
            ( intData[max(1,min(x+xMaxCurr+1,h))*(w+1) 
                + max(1,min(y+yMaxCurr,w))]
            - intData[max(1,min(x+xMaxCurr,h))*(w+1)
                + max(1,min(y+yMaxCurr,w))]
            - intData[max(1,min(x+xMaxCurr+1,h))*(w+1)
                + max(0,min(y+yMinCurr,w-1))]
            + intData[max(1,min(x+xMaxCurr,h))*(w+1)
                + max(0,min(y+yMinCurr,w-1))]
            ) * xMaxCurrFrac;

        // -- yMax border
        outValue +=
            ( intData[max(1,min(x+xMaxCurr,h))*(w+1) 
                + max(1,min(y+yMaxCurr+1,w))]
            - intData[max(1,min(x+xMaxCurr,h))*(w+1)
                + max(1,min(y+yMaxCurr,w))]
            - intData[max(0,min(x+xMinCurr,h-1))*(w+1)
                + max(1,min(y+yMaxCurr+1,w))]
            + intData[max(0,min(x+xMinCurr,h-1))*(w+1)
                + max(1,min(y+yMaxCurr,w))]
            ) * yMaxCurrFrac;

        // -- xMin border
        outValue +=
            ( intData[max(0,min(x+xMinCurr,h-1))*(w+1) 
                + max(1,min(y+yMaxCurr,w))]
            - intData[max(0,min(x+xMinCurr-1,h-1))*(w+1)
                + max(1,min(y+yMaxCurr,w))]
            - intData[max(0,min(x+xMinCurr,h-1))*(w+1)
                + max(0,min(y+yMinCurr,w-1))]
            + intData[max(0,min(x+xMinCurr-1,h-1))*(w+1)
                + max(0,min(y+yMinCurr,w-1))]
            ) * xMinCurrFrac;

        // -- yMin border
        outValue +=
            ( intData[max(1,min(x+xMaxCurr,h))*(w+1) 
                + max(0,min(y+yMinCurr,w-1))]
            - intData[max(1,min(x+xMaxCurr,h))*(w+1)
                + max(0,min(y+yMinCurr-1,w-1))]
            - intData[max(0,min(x+xMinCurr,h-1))*(w+1)
                + max(0,min(y+yMinCurr,w-1))]
            + intData[max(0,min(x+xMinCurr,h-1))*(w+1)
                + max(0,min(y+yMinCurr-1,w-1))]
            ) * yMinCurrFrac;

        // -- corner pixels
        outValue += 
            xMaxCurrFrac*yMaxCurrFrac * (
               (x+xMaxCurr >  h-1 or
                y+yMaxCurr >  w-1 or
                x+xMaxCurr <= 0   or
                y+yMaxCurr <= 0) ? 0 : inData[(x+xMaxCurr)*inDataStrideRow + (y+yMaxCurr)]);

        outValue +=
            xMinCurrFrac*yMaxCurrFrac * (
               (x+xMinCurr-1 >= h-1 or
                y+yMaxCurr   >  w-1 or
                x+xMinCurr-1 <  0   or
                y+yMaxCurr   <= 0) ? 0 : inData[(x+xMinCurr-1)*inDataStrideRow + (y+yMaxCurr)]);

        outValue +=
            xMaxCurrFrac*yMinCurrFrac * (
               (x+xMaxCurr   >  h-1 or
                y+yMinCurr-1 >= w-1 or
                x+xMaxCurr   <= 0   or
                y+yMinCurr-1 <  0) ? 0 : inData[(x+xMaxCurr)*inDataStrideRow + (y+yMinCurr-1)]);

        outValue +=
            xMinCurrFrac*yMinCurrFrac * (
               (x+xMinCurr-1 >= h-1 or
                y+yMinCurr-1 >= w-1 or
                x+xMinCurr-1 <  0   or
                y+yMinCurr-1 <  0) ? 0 : inData[(x+xMinCurr-1)*inDataStrideRow + (y+yMinCurr-1)]);

        outData[z*w*h + x*w + y] = outValue;
    }
}

extern "C" {

void forwardCuda(THCState *state,
    float *intData, int h, int w, int nWindows, float *outData,
    float *xMin, float *xMax, float *yMin, float *yMax, float *areaCoeff) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_CHANNELS);
    dim3 dimGrid((h + dimBlock.x - 1) / dimBlock.x, (w + dimBlock.y - 1) / dimBlock.y, (nWindows + dimBlock.z - 1) / dimBlock.z);

    forwardKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> 
        (intData, outData, h, w, nWindows, xMin, xMax, yMin, yMax, areaCoeff);
    THCudaCheck(hipGetLastError());
}

void forwardNoNormReplicateCuda(THCState *state,
    float *intData, int intDataStrideChannel, float *outData,
    int h, int w, int nInputPlane, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        strided::forwardNoNormReplicateCuda(
            intData, intDataStrideChannel, outData,
            h, w, nInputPlane, nWindows,
            xMin, xMax, yMin, yMax,
            strideH, strideW);
        return;
    }

    // TODO: 1D grid
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_CHANNELS);
    dim3 dimGrid(
        (h + dimBlock.x - 1) / dimBlock.x, 
        (w + dimBlock.y - 1) / dimBlock.y, 
        (nInputPlane*nWindows + dimBlock.z - 1) / dimBlock.z);

    forwardNoNormReplicateKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, intDataStrideChannel, outData,
        h, w, nInputPlane, nWindows,
        xMin, xMax, yMin, yMax);
    THCudaCheck(hipGetLastError());
}

void forwardNoNormReplicateFracCuda(THCState *state,
    float *intData, int intDataStrideChannel, float *outData,
    int h, int w, int nInputPlane, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax,
    float *inData, int inDataStrideRow, int inDataStrideChannel,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        strided::forwardNoNormReplicateFracCuda(
            intData, intDataStrideChannel, outData,
            h, w, nInputPlane, nWindows,
            xMin, xMax, yMin, yMax,
            inData, inDataStrideRow, inDataStrideChannel,
            strideH, strideW);
        return;
    }

    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid((nInputPlane*nWindows*h*w + dimBlock.x - 1) / dimBlock.x);

    forwardNoNormReplicateFracKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, intDataStrideChannel, outData,
        h, w, nInputPlane, nWindows, 
        xMin, xMax, yMin, yMax,
        inData, inDataStrideRow, inDataStrideChannel);
    THCudaCheck(hipGetLastError());
}

/************************ updateGradInput ************************/

/************** Planewise *************/

__global__ void updateGradInputPlanewiseKernel(
    float *gradOutputIntData, float *gradInputData,
    int h, int w, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax) {

    const int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    if (x < h and y < w) {

        int xMinCurr, xMaxCurr, yMinCurr, yMaxCurr;
        double outValue = 0;

        for (int windowIdx = 0; windowIdx < nWindows; ++windowIdx) {

            xMinCurr = (int)ceil(-xMax[windowIdx]);
            yMinCurr = (int)ceil(-yMax[windowIdx]);

            xMaxCurr = (int)floor(-xMin[windowIdx]) + 1;
            yMaxCurr = (int)floor(-yMin[windowIdx]) + 1;

            // The following code block implements these lines
            // as if they were executed simultaneously (see `void updateGradInputFrac()`):
            // xMinCurr = (x == 0   and xMaxCurr >= 0 ? 0    : xMinCurr);
            // xMaxCurr = (x == h-1 and xMinCurr <= 0 ? h+66 : xMaxCurr);
            // yMinCurr = (y == 0   and yMaxCurr >= 0 ? 0    : yMinCurr);
            // yMaxCurr = (y == w-1 and yMinCurr <= 0 ? w+66 : yMaxCurr);

            bool needToChangeMin, needToChangeMax;

            needToChangeMin = x == 0   and xMaxCurr >= 0;
            needToChangeMax = x == h-1 and xMinCurr <= 0;
            if (needToChangeMin) xMinCurr = 0;
            if (needToChangeMax) xMaxCurr = h+66;

            needToChangeMin = y == 0   and yMaxCurr >= 0;
            needToChangeMax = y == w-1 and yMinCurr <= 0;
            if (needToChangeMin) yMinCurr = 0;
            if (needToChangeMax) yMaxCurr = w+66;

            const int t = max(0, min(x+xMinCurr, h) );
            const int b = max(0, min(x+xMaxCurr, h) );
            const int l = max(0, min(y+yMinCurr, w) );
            const int r = max(0, min(y+yMaxCurr, w) );

            outValue += gradOutputIntData[b*(w+1) + r];
            outValue -= gradOutputIntData[t*(w+1) + r];
            outValue -= gradOutputIntData[b*(w+1) + l];
            outValue += gradOutputIntData[t*(w+1) + l];

            // go to the next channel
            gradOutputIntData += (h+1)*(w+1);
        }

        gradInputData[x*w + y] = outValue;
    }
}

__global__ void updateGradInputPlanewiseFracKernel(
    float *gradOutputIntData, float *gradInputData,
    int h, int w, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax,
    float *gradOutputData, int gradOutputStrideRow, int gradOutputStrideChannel) {

    const int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    if (x < h and y < w) {

        int xMinCurr, xMaxCurr, yMinCurr, yMaxCurr;
        double outValue = 0;

        for (int windowIdx = 0; windowIdx < nWindows; ++windowIdx) {

            xMinCurr = (int)ceil(-xMax[windowIdx]);
            yMinCurr = (int)ceil(-yMax[windowIdx]);
            const float xMinCurrFrac = (float)xMinCurr + xMax[windowIdx];
            const float yMinCurrFrac = (float)yMinCurr + yMax[windowIdx];

            xMaxCurr = (int)floor(-xMin[windowIdx]) + 1;
            yMaxCurr = (int)floor(-yMin[windowIdx]) + 1;
            const float xMaxCurrFrac = -xMin[windowIdx] + 1 - xMaxCurr;
            const float yMaxCurrFrac = -yMin[windowIdx] + 1 - yMaxCurr;

            // The following code block implements these lines
            // as if they were executed simultaneously (see `void updateGradInputFrac()`):
            // xMinCurr = (x == 0   and xMaxCurr >= 0 ? 0    : xMinCurr);
            // xMaxCurr = (x == h-1 and xMinCurr <= 0 ? h+66 : xMaxCurr);
            // yMinCurr = (y == 0   and yMaxCurr >= 0 ? 0    : yMinCurr);
            // yMaxCurr = (y == w-1 and yMinCurr <= 0 ? w+66 : yMaxCurr);

            bool needToChangeMin, needToChangeMax;

            needToChangeMin = x == 0   and xMaxCurr >= 0;
            needToChangeMax = x == h-1 and xMinCurr <= 0;
            if (needToChangeMin) xMinCurr = 0;
            if (needToChangeMax) xMaxCurr = h+66;

            needToChangeMin = y == 0   and yMaxCurr >= 0;
            needToChangeMax = y == w-1 and yMinCurr <= 0;
            if (needToChangeMin) yMinCurr = 0;
            if (needToChangeMax) yMaxCurr = w+66;

            const int t = max(0, min(x+xMinCurr, h) );
            const int b = max(0, min(x+xMaxCurr, h) );
            const int l = max(0, min(y+yMinCurr, w) );
            const int r = max(0, min(y+yMaxCurr, w) );

            const int tAdv = x+xMinCurr-1 <  h ? max(0, min(t-1, h)) : t;
            const int bAdv = x+xMaxCurr   >= 0 ? max(0, min(b+1, h)) : b;
            const int lAdv = y+yMinCurr-1 <  w ? max(0, min(l-1, w)) : l;
            const int rAdv = y+yMaxCurr   >= 0 ? max(0, min(r+1, w)) : r;

            // TODO: 1D grid
            outValue += gradOutputIntData[b*(w+1) + r];
            outValue -= gradOutputIntData[t*(w+1) + r];
            outValue -= gradOutputIntData[b*(w+1) + l];
            outValue += gradOutputIntData[t*(w+1) + l];

            // -- xMax border
            outValue +=
                ( gradOutputIntData[bAdv*(w+1) + r]
                - gradOutputIntData[b   *(w+1) + r]
                - gradOutputIntData[bAdv*(w+1) + l]
                + gradOutputIntData[b   *(w+1) + l]
                ) * xMaxCurrFrac;

            // -- yMax border
            outValue +=
                ( gradOutputIntData[b*(w+1) + rAdv]
                - gradOutputIntData[b*(w+1) + r   ]
                - gradOutputIntData[t*(w+1) + rAdv]
                + gradOutputIntData[t*(w+1) + r   ]
                ) * yMaxCurrFrac;

            // -- xMin border
            outValue +=
                ( gradOutputIntData[t   *(w+1) + r]
                - gradOutputIntData[tAdv*(w+1) + r]
                - gradOutputIntData[t   *(w+1) + l]
                + gradOutputIntData[tAdv*(w+1) + l]
                ) * xMinCurrFrac;

            // -- yMin border
            outValue +=
                ( gradOutputIntData[b*(w+1) + l   ]
                - gradOutputIntData[b*(w+1) + lAdv]
                - gradOutputIntData[t*(w+1) + l   ]
                + gradOutputIntData[t*(w+1) + lAdv]
                ) * yMinCurrFrac;

            // -- corner pixels
            outValue += 
                xMaxCurrFrac*yMaxCurrFrac * (
                   (x+xMaxCurr > h-1 or
                    y+yMaxCurr > w-1 or
                    x+xMaxCurr < 0   or
                    y+yMaxCurr < 0   or
                    b == bAdv or
                    r == rAdv) ? 0 : 
                    gradOutputData[b*gradOutputStrideRow + r]);

            outValue +=
                xMinCurrFrac*yMaxCurrFrac * (
                   (x+xMinCurr-1 > h-1 or
                    y+yMaxCurr   > w-1 or
                    x+xMinCurr-1 < 0   or
                    y+yMaxCurr   < 0   or
                    t == tAdv or
                    r == rAdv) ? 0 : 
                    gradOutputData[tAdv*gradOutputStrideRow + r]);

            outValue +=
                xMaxCurrFrac*yMinCurrFrac * (
                   (x+xMaxCurr   > h-1 or
                    y+yMinCurr-1 > w-1 or
                    x+xMaxCurr   < 0   or
                    y+yMinCurr-1 < 0   or
                    b == bAdv or
                    l == lAdv) ? 0 : 
                    gradOutputData[b*gradOutputStrideRow + lAdv]);

            outValue +=
                xMinCurrFrac*yMinCurrFrac * (
                   (x+xMinCurr-1 > h-1 or
                    y+yMinCurr-1 > w-1 or
                    x+xMinCurr-1 < 0   or
                    y+yMinCurr-1 < 0   or
                    t == tAdv or
                    l == lAdv) ? 0 : 
                    gradOutputData[tAdv*gradOutputStrideRow + lAdv]);

            // go to the next channel
            gradOutputIntData += (h+1)*(w+1);
            gradOutputData += gradOutputStrideChannel;
        }

        gradInputData[x*w + y] = outValue;
    }
}

void updateGradInputPlanewiseCuda(THCState *state,
    float *gradOutputIntData, float *gradInputData,
    int h, int w, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        strided::updateGradInputPlanewiseCuda(
            gradOutputIntData, gradInputData, h, w, nWindows,
            xMin, xMax, yMin, yMax, strideH, strideW);
        return;
    }

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_CHANNELS);
    dim3 dimGrid(
        (h + dimBlock.x - 1) / dimBlock.x, 
        (w + dimBlock.y - 1) / dimBlock.y);

    updateGradInputPlanewiseKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        gradOutputIntData, gradInputData,
        h, w, nWindows,
        xMin, xMax, yMin, yMax);
    THCudaCheck(hipGetLastError());
}

void updateGradInputPlanewiseFracCuda(THCState *state,
    float *gradOutputIntData, float *gradInputData,
    int h, int w, int nWindows,
    float *xMin, float *xMax, float *yMin, float *yMax,
    float *gradOutputData, int gradOutputStrideRow, int gradOutputStrideChannel,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        strided::updateGradInputPlanewiseFracCuda(
            gradOutputIntData, gradInputData, h, w, nWindows,
            xMin, xMax, yMin, yMax,
            gradOutputData, gradOutputStrideRow, gradOutputStrideChannel,
            strideH, strideW);
        return;
    }

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_CHANNELS);
    dim3 dimGrid(
        (h + dimBlock.x - 1) / dimBlock.x, 
        (w + dimBlock.y - 1) / dimBlock.y);

    updateGradInputPlanewiseFracKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        gradOutputIntData, gradInputData,
        h, w, nWindows,
        xMin, xMax, yMin, yMax,
        gradOutputData, gradOutputStrideRow, gradOutputStrideChannel);
    THCudaCheck(hipGetLastError());
}

/****************** Single-kernel updateGradInput (faster) **************/

__global__ void updateGradInputKernel(
    const float *gradOutputIntData, float *tmpArray,
    const int batchSize, const int nInputPlane, const int nWindows,
    const int h, const int w,
    const float *const xMin, const float *const xMax,
    const float *const yMin, const float *const yMax) {

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w; id /= w;
    const int x = id % h; id /= h;
    const int globalWindowIdx = id % (nInputPlane * nWindows);

    // `id` is now the current plane number
    gradOutputIntData += id * (w+1) * (h+1);

    if (id < batchSize * nInputPlane * nWindows) {

        float outValue = 0;
        int xMinCurr, xMaxCurr, yMinCurr, yMaxCurr;

        xMinCurr = (int)ceil(-xMax[globalWindowIdx]);
        yMinCurr = (int)ceil(-yMax[globalWindowIdx]);

        xMaxCurr = (int)floor(-xMin[globalWindowIdx]) + 1;
        yMaxCurr = (int)floor(-yMin[globalWindowIdx]) + 1;

        // The following code block implements these lines
        // as if they were executed simultaneously (see `void updateGradInputFrac()`):
        // xMinCurr = (x == 0   and xMaxCurr >= 0 ? 0    : xMinCurr);
        // xMaxCurr = (x == h-1 and xMinCurr <= 0 ? h+66 : xMaxCurr);
        // yMinCurr = (y == 0   and yMaxCurr >= 0 ? 0    : yMinCurr);
        // yMaxCurr = (y == w-1 and yMinCurr <= 0 ? w+66 : yMaxCurr);

        bool needToChangeMin, needToChangeMax;

        needToChangeMin = x == 0   and xMaxCurr >= 0;
        needToChangeMax = x == h-1 and xMinCurr <= 0;
        if (needToChangeMin) xMinCurr = 0;
        if (needToChangeMax) xMaxCurr = h+66;

        needToChangeMin = y == 0   and yMaxCurr >= 0;
        needToChangeMax = y == w-1 and yMinCurr <= 0;
        if (needToChangeMin) yMinCurr = 0;
        if (needToChangeMax) yMaxCurr = w+66;

        const int t = max(0, min(x+xMinCurr, h) );
        const int b = max(0, min(x+xMaxCurr, h) );
        const int l = max(0, min(y+yMinCurr, w) );
        const int r = max(0, min(y+yMaxCurr, w) );

        outValue += gradOutputIntData[b*(w+1) + r];
        outValue -= gradOutputIntData[t*(w+1) + r];
        outValue -= gradOutputIntData[b*(w+1) + l];
        outValue += gradOutputIntData[t*(w+1) + l];

        *tmpArray = outValue;
    }
}

__global__ void updateGradInputFracKernel(
    const float *gradOutputIntData, float *tmpArray,
    const int batchSize, const int nInputPlane, const int nWindows,
    const int h, const int w,
    const float *const xMin, const float *const xMax,
    const float *const yMin, const float *const yMax,
    const float *gradOutputData,
    const int gradOutputStrideRow, const int gradOutputStrideChannel) {

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w; id /= w;
    const int x = id % h; id /= h;
    const int globalWindowIdx = id % (nInputPlane * nWindows);

    // `id` is now the current plane number
    gradOutputIntData += id * (w+1) * (h+1);
    gradOutputData += id * gradOutputStrideChannel;

    if (id < batchSize * nInputPlane * nWindows) {

        double outValue = 0;
        int xMinCurr, xMaxCurr, yMinCurr, yMaxCurr;

        xMinCurr = (int)ceil(-xMax[globalWindowIdx]);
        yMinCurr = (int)ceil(-yMax[globalWindowIdx]);
        const float xMinCurrFrac = (float)xMinCurr + xMax[globalWindowIdx];
        const float yMinCurrFrac = (float)yMinCurr + yMax[globalWindowIdx];

        xMaxCurr = (int)floor(-xMin[globalWindowIdx]) + 1;
        yMaxCurr = (int)floor(-yMin[globalWindowIdx]) + 1;
        const float xMaxCurrFrac = -xMin[globalWindowIdx] + 1 - xMaxCurr;
        const float yMaxCurrFrac = -yMin[globalWindowIdx] + 1 - yMaxCurr;

        // The following code block implements these lines
        // as if they were executed simultaneously (see `void updateGradInputFrac()`):
        // xMinCurr = (x == 0   and xMaxCurr >= 0 ? 0    : xMinCurr);
        // xMaxCurr = (x == h-1 and xMinCurr <= 0 ? h+66 : xMaxCurr);
        // yMinCurr = (y == 0   and yMaxCurr >= 0 ? 0    : yMinCurr);
        // yMaxCurr = (y == w-1 and yMinCurr <= 0 ? w+66 : yMaxCurr);

        bool needToChangeMin, needToChangeMax;

        needToChangeMin = x == 0   and xMaxCurr >= 0;
        needToChangeMax = x == h-1 and xMinCurr <= 0;
        if (needToChangeMin) xMinCurr = 0;
        if (needToChangeMax) xMaxCurr = h+66;

        needToChangeMin = y == 0   and yMaxCurr >= 0;
        needToChangeMax = y == w-1 and yMinCurr <= 0;
        if (needToChangeMin) yMinCurr = 0;
        if (needToChangeMax) yMaxCurr = w+66;

        const int t = max(0, min(x+xMinCurr, h) );
        const int b = max(0, min(x+xMaxCurr, h) );
        const int l = max(0, min(y+yMinCurr, w) );
        const int r = max(0, min(y+yMaxCurr, w) );

        const int tAdv = x+xMinCurr-1 <  h ? max(0, min(t-1, h)) : t;
        const int bAdv = x+xMaxCurr   >= 0 ? max(0, min(b+1, h)) : b;
        const int lAdv = y+yMinCurr-1 <  w ? max(0, min(l-1, w)) : l;
        const int rAdv = y+yMaxCurr   >= 0 ? max(0, min(r+1, w)) : r;

        outValue += gradOutputIntData[b*(w+1) + r];
        outValue -= gradOutputIntData[t*(w+1) + r];
        outValue -= gradOutputIntData[b*(w+1) + l];
        outValue += gradOutputIntData[t*(w+1) + l];

        // -- xMax border
        outValue +=
            ( gradOutputIntData[bAdv*(w+1) + r]
            - gradOutputIntData[b   *(w+1) + r]
            - gradOutputIntData[bAdv*(w+1) + l]
            + gradOutputIntData[b   *(w+1) + l]
            ) * xMaxCurrFrac;

        // -- yMax border
        outValue +=
            ( gradOutputIntData[b*(w+1) + rAdv]
            - gradOutputIntData[b*(w+1) + r   ]
            - gradOutputIntData[t*(w+1) + rAdv]
            + gradOutputIntData[t*(w+1) + r   ]
            ) * yMaxCurrFrac;

        // -- xMin border
        outValue +=
            ( gradOutputIntData[t   *(w+1) + r]
            - gradOutputIntData[tAdv*(w+1) + r]
            - gradOutputIntData[t   *(w+1) + l]
            + gradOutputIntData[tAdv*(w+1) + l]
            ) * xMinCurrFrac;

        // -- yMin border
        outValue +=
            ( gradOutputIntData[b*(w+1) + l   ]
            - gradOutputIntData[b*(w+1) + lAdv]
            - gradOutputIntData[t*(w+1) + l   ]
            + gradOutputIntData[t*(w+1) + lAdv]
            ) * yMinCurrFrac;

        // -- corner pixels
        outValue += 
            xMaxCurrFrac*yMaxCurrFrac * (
               (x+xMaxCurr > h-1 or
                y+yMaxCurr > w-1 or
                x+xMaxCurr < 0   or
                y+yMaxCurr < 0   or
                b == bAdv or
                r == rAdv) ? 0 : 
                gradOutputData[b*gradOutputStrideRow + r]);

        outValue +=
            xMinCurrFrac*yMaxCurrFrac * (
               (x+xMinCurr-1 > h-1 or
                y+yMaxCurr   > w-1 or
                x+xMinCurr-1 < 0   or
                y+yMaxCurr   < 0   or
                t == tAdv or
                r == rAdv) ? 0 : 
                gradOutputData[tAdv*gradOutputStrideRow + r]);

        outValue +=
            xMaxCurrFrac*yMinCurrFrac * (
               (x+xMaxCurr   > h-1 or
                y+yMinCurr-1 > w-1 or
                x+xMaxCurr   < 0   or
                y+yMinCurr-1 < 0   or
                b == bAdv or
                l == lAdv) ? 0 : 
                gradOutputData[b*gradOutputStrideRow + lAdv]);

        outValue +=
            xMinCurrFrac*yMinCurrFrac * (
               (x+xMinCurr-1 > h-1 or
                y+yMinCurr-1 > w-1 or
                x+xMinCurr-1 < 0   or
                y+yMinCurr-1 < 0   or
                t == tAdv or
                l == lAdv) ? 0 : 
                gradOutputData[tAdv*gradOutputStrideRow + lAdv]);

        *tmpArray = outValue;
    }
}

void updateGradInputCuda(THCState *state,
    const float *gradOutputIntData, float *tmpArray,
    const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w,
    const float *const xMin, const float *const xMax,
    const float *const yMin, const float *const yMax,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        THError("NYI");
        // strided::updateGradInputPlanewiseFracCuda(
        //     gradOutputIntData, gradInputData, h, w, nWindows,
        //     xMin, xMax, yMin, yMax,
        //     gradOutputData, gradOutputStrideRow, gradOutputStrideChannel,
        //     strideH, strideW);
        return;
    }

    const int NUM_THREADS = BLOCK_SIZE * BLOCK_SIZE;
    const int threadsNeeded = batchSize * nInputPlane * nWindows * h * w;
    const int numBlocks = (threadsNeeded + NUM_THREADS - 1) / NUM_THREADS;

    updateGradInputKernel <<<numBlocks, NUM_THREADS, 0, THCState_getCurrentStream(state)>>> (
        gradOutputIntData, tmpArray,
        batchSize, nInputPlane, nWindows,
        h, w, xMin, xMax, yMin, yMax);
    THCudaCheck(hipGetLastError());
}

void updateGradInputFracCuda(THCState *state,
    const float *gradOutputIntData, float *tmpArray,
    const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w,
    const float *const xMin, const float *const xMax,
    const float *const yMin, const float *const yMax,
    const float *gradOutputData,
    const int gradOutputStrideRow, const int gradOutputStrideChannel,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        THError("NYI");
        // strided::updateGradInputPlanewiseFracCuda(
        //     gradOutputIntData, gradInputData, h, w, nWindows,
        //     xMin, xMax, yMin, yMax,
        //     gradOutputData, gradOutputStrideRow, gradOutputStrideChannel,
        //     strideH, strideW);
        return;
    }

    const int NUM_THREADS = BLOCK_SIZE * BLOCK_SIZE;
    const int threadsNeeded = batchSize * nInputPlane * nWindows * h * w;
    const int numBlocks = (threadsNeeded + NUM_THREADS - 1) / NUM_THREADS;

    updateGradInputFracKernel <<<numBlocks, NUM_THREADS, 0, THCState_getCurrentStream(state)>>> (
        gradOutputIntData, tmpArray,
        batchSize, nInputPlane, nWindows,
        h, w, xMin, xMax, yMin, yMax,
        gradOutputData, gradOutputStrideRow, gradOutputStrideChannel);
    THCudaCheck(hipGetLastError());
}

/************************ accGradParameters ************************/

__global__ void xMaxDeltaIntegralFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMax, const float *yMin, const float *yMax,
    const float *inData, const int inDataStrideRow) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        // const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        // const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        const int xMaxInt = (int)floor(xMax[windowIdx]);
        // const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        const int yMaxInt = (int)floor(yMax[windowIdx]);
        const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        // const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        // const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];
        const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += brCorner * (y+yMaxInt <  1 ? 1.0f : yMaxFrac);
        delta += blCorner * (y+yMinInt >= w ? 1.0f : yMinFrac);

        delta += 
            intData[max(0,min(x+xMaxInt+1, h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMaxInt  , h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMaxInt+1, h))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMaxInt  , h))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMaxInt >= 1 and x+xMaxInt < h);
        tmpArray[(x-1)*w + (y-1)] *= delta;
    }
}

__global__ void xMinDeltaIntegralFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *yMin, const float *yMax,
    const float *inData, const int inDataStrideRow) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        // const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        // const int xMaxInt = (int)floor(xMax[windowIdx]);
        // const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        const int yMaxInt = (int)floor(yMax[windowIdx]);
        const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        // const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];
        // const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += trCorner * (y+yMaxInt <  1 ? 1.0f : yMaxFrac);
        delta += tlCorner * (y+yMinInt >= w ? 1.0f : yMinFrac);

        delta += 
            intData[max(0,min(x+xMinInt  , h))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt-1, h))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt  , h))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMinInt-1, h))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMinInt >= 1 and x+xMinInt < h);
        tmpArray[(x-1)*w + (y-1)] *= -delta;
    }
}

__global__ void yMaxDeltaIntegralFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMax,
    const float *inData, const int inDataStrideRow) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        // const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        // const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        const int yMaxInt = (int)floor(yMax[windowIdx]);
        // const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        // const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        // const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMinInt-1))];
        const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];
        const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += trCorner * (x+xMinInt >= h ? 1.0f : xMinFrac);
        delta += brCorner * (x+xMaxInt <  1 ? 1.0f : xMaxFrac);

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMaxInt+1, w))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMaxInt+1, w))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];

        delta *= (y+yMaxInt >= 1 and y+yMaxInt < w);
        tmpArray[(x-1)*w + (y-1)] *= delta;
    }
}

__global__ void yMinDeltaIntegralFracKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMin,
    const float *inData, const int inDataStrideRow) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const float xMinFrac = xMinInt-xMin[windowIdx]+1;

        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        // const float yMinFrac = yMinInt-yMin[windowIdx]+1;

        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const float xMaxFrac = xMax[windowIdx]-xMaxInt;

        // const int yMaxInt = (int)floor(yMax[windowIdx]);
        // const float yMaxFrac = yMax[windowIdx]-yMaxInt;

        const float tlCorner = y+yMinInt <  1 or x+xMinInt <  1 ? 0 :
                             inData[
                                max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        const float blCorner = y+yMinInt <  1 or x+xMaxInt >= h ? 0 :
                            inData[
                                max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
                                max(0,min(w-1,y+yMinInt-1))];
        // const float trCorner = y+yMaxInt >= w or x+xMinInt <  1 ? 0 :
        //                      inData[
        //                         max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];
        // const float brCorner = y+yMaxInt >= w or x+xMaxInt >= h ? 0 :
        //                     inData[
        //                         max(0,min(h-1,x+xMaxInt  )) * inDataStrideRow +
        //                         max(0,min(w-1,y+yMaxInt  ))];

        float delta = 0;

        delta += tlCorner * (x+xMinInt >= h ? 1.0f : xMinFrac);
        delta += blCorner * (x+xMaxInt <  1 ? 1.0f : xMaxFrac);

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w))];

        delta *= (y+yMinInt >= 1 and y+yMinInt < w);
        tmpArray[(x-1)*w + (y-1)] *= -delta;
    }
}

void backwardFracCuda(THCState *state,
    float *intData, float *tmpArray,
    int nWindows, int h, int w,
    float *xMin, float *xMax, float *yMin, float *yMax,
    float *inData, int inDataStrideRow,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        strided::backwardFracCuda(
            intData, tmpArray, nWindows, h, w,
            xMin, xMax, yMin, yMax, inData, inDataStrideRow,
            strideH, strideW);
        return;
    }

    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid((nWindows * h * w + dimBlock.x - 1) / dimBlock.x);

    xMaxDeltaIntegralFracKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 0*nWindows*h*w, nWindows, h, w,
        xMax, yMin, yMax, inData, inDataStrideRow);

    xMinDeltaIntegralFracKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 1*nWindows*h*w, nWindows, h, w,
        xMin, yMin, yMax, inData, inDataStrideRow);

    yMaxDeltaIntegralFracKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 2*nWindows*h*w, nWindows, h, w,
        xMin, xMax, yMax, inData, inDataStrideRow);

    yMinDeltaIntegralFracKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 3*nWindows*h*w, nWindows, h, w,
        xMin, xMax, yMin, inData, inDataStrideRow);
    THCudaCheck(hipGetLastError());
}

__global__ void xMaxDeltaIntegralKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMax, const float *yMin, const float *yMax) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        // const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(1,min(x+xMaxInt+1, h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMaxInt  , h))*(w+1) 
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(1,min(x+xMaxInt+1, h))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMaxInt  , h))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMaxInt >= 1 and x+xMaxInt < h);
        tmpArray[(x-1)*w + (y-1)] *= delta;
    }
}

__global__ void xMinDeltaIntegralKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *yMin, const float *yMax) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        // const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(0,min(x+xMinInt  , h-1))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt-1, h  ))*(w+1)
                  + max(0,min(y+yMaxInt, w))];
        delta -=
            intData[max(0,min(x+xMinInt  , h-1))*(w+1)
                  + max(0,min(y+yMinInt, w))];
        delta +=
            intData[max(0,min(x+xMinInt-1, h  ))*(w+1)
                  + max(0,min(y+yMinInt, w))];

        delta *= (x+xMinInt >= 1 and x+xMinInt < h);
        tmpArray[(x-1)*w + (y-1)] *= -delta;
    }
}

__global__ void yMaxDeltaIntegralKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMax) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        // const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(1,min(y+yMaxInt+1, w))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(1,min(y+yMaxInt+1, w))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMaxInt  , w))];

        delta *= (y+yMaxInt >= 1 and y+yMaxInt < w);
        tmpArray[(x-1)*w + (y-1)] *= delta;
    }
}

__global__ void yMinDeltaIntegralKernel(
    const float *intData, float *tmpArray,
    const int nWindows, const int h, const int w,
    const float *xMin, const float *xMax, const float *yMin) {
 
    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const int yMinInt = (int)ceil(yMin[windowIdx]-1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        // const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        delta += 
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w  ))];
        delta -=
            intData[max(0,min(x+xMaxInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w-1))];
        delta -=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt  , w  ))];
        delta +=
            intData[max(0,min(x+xMinInt, h))*(w+1)
                  + max(0,min(y+yMinInt-1, w-1))];

        delta *= (y+yMinInt >= 1 and y+yMinInt < w);
        tmpArray[(x-1)*w + (y-1)] *= -delta;
    }
}

void backwardCuda(THCState *state,
    float *intData, float *tmpArray,
    int nWindows, int h, int w,
    float *xMin, float *xMax, float *yMin, float *yMax,
    const int strideH, const int strideW) {

    if (strideH != 1 or strideW != 1) {
        strided::backwardCuda(
            intData, tmpArray, nWindows, h, w,
            xMin, xMax, yMin, yMax, strideH, strideW);
        return;
    }

    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid((nWindows * h * w + dimBlock.x - 1) / dimBlock.x);

    xMaxDeltaIntegralKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 0*nWindows*h*w,
        nWindows, h, w, xMax, yMin, yMax);

    xMinDeltaIntegralKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 1*nWindows*h*w,
        nWindows, h, w, xMin, yMin, yMax);

    yMaxDeltaIntegralKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 2*nWindows*h*w,
        nWindows, h, w, xMin, xMax, yMax);

    yMinDeltaIntegralKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        intData, tmpArray + 3*nWindows*h*w,
        nWindows, h, w, xMin, xMax, yMin);
    THCudaCheck(hipGetLastError());
}

/************************ Other stuff ************************/

__global__ void dirtyFixWindowsKernel(
    float *xMin, float *xMax, float *yMin, float *yMax,
    const int size, const float h, const float w, const float minWidth) {

    int idx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

    if (idx < 2*size) {
        float paramMin, paramMax;

        if (idx < size) {
            paramMin = max(-h+1, min(h-1, xMin[idx]));
            paramMax = max(-h+1, min(h-1, xMax[idx]));

            if (paramMin + minWidth - 0.99 > paramMax) {
                const float mean = 0.5 * (paramMin + paramMax);
                paramMin = mean - 0.5 * (minWidth - 0.9);
                paramMax = mean + 0.5 * (minWidth - 0.9);
            }

            xMin[idx] = paramMin;
            xMax[idx] = paramMax;
        } else {
            idx -= size;
            paramMin = max(-w+1, min(w-1, yMin[idx]));
            paramMax = max(-w+1, min(w-1, yMax[idx]));

            if (paramMin + minWidth - 0.99 > paramMax) {
                const float mean = 0.5 * (paramMin + paramMax);
                paramMin = mean - 0.5 * (minWidth - 0.9);
                paramMax = mean + 0.5 * (minWidth - 0.9);
            }

            yMin[idx] = paramMin;
            yMax[idx] = paramMax;
        }
    }
}

void dirtyFixWindows(THCState *state,
    float *xMin, float *xMax, float *yMin, float *yMax,
    int size, int h, int w, float minWidth) {

    dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
    dim3 dimGrid((2*size + dimBlock.x - 1) / dimBlock.x);

    dirtyFixWindowsKernel <<<dimGrid, dimBlock, 0, THCState_getCurrentStream(state)>>> (
        xMin, xMax, yMin, yMax, size, (float)h, (float)w, minWidth);
    THCudaCheck(hipGetLastError());
}

} // extern "C"
