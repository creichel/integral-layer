
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <algorithm>

#define BLOCK_SIZE 32

using std::max;
using std::min;

__global__ void forwardKernel(
    float *intData, float *outData, int h, int w, int xMinCurr, 
    int xMaxCurr, int yMinCurr, int yMaxCurr, float areaCoeff) {

    int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    if (x < h and y < w) {
        int t = max(0, min(x+xMinCurr, h) );
        int b = max(0, min(x+xMaxCurr, h) );
        int l = max(0, min(y+yMinCurr, w) );
        int r = max(0, min(y+yMaxCurr, w) );

        outData[x*w + y] = areaCoeff *
            ( intData[b*(w+1) + r]
            - intData[t*(w+1) + r]
            - intData[b*(w+1) + l]
            + intData[t*(w+1) + l]);
    }
}

extern "C" {

void forwardCuda(
    float *intData, int h, int w, float *outData,
    int xMinCurr, int xMaxCurr, int yMinCurr, int yMaxCurr, float areaCoeff) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((h + dimBlock.x - 1) / dimBlock.x, (w + dimBlock.y - 1) / dimBlock.y);

    forwardKernel <<<dimGrid, dimBlock>>> (intData, outData, h, w, xMinCurr, xMaxCurr, yMinCurr, yMaxCurr, areaCoeff);
}

void backward(
    float *intData, float *gradOutData, int h, int w, float *deltas,
    int xMinCurr, int xMaxCurr, int yMinCurr, int yMaxCurr) {

    float & xMaxDelta = deltas[1];
    float & xMinDelta = deltas[0];
    float & yMaxDelta = deltas[3];
    float & yMinDelta = deltas[2];

    for (int x = 1; x <= h; ++x) {
        for (int y = 1; y <= w; ++y) {
            
            int tClip = max(x+xMinCurr, 0);
            int bClip = min(x+xMaxCurr, h);
            int lClip = max(y+yMinCurr, 0);
            int rClip = min(y+yMaxCurr, w);

            xMaxDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,min(x+xMaxCurr+1,h))*(w+1) + max(0,rClip)]
                - intData[max(0,bClip)*(w+1) + max(0,rClip)]
                - intData[max(0,min(x+xMaxCurr+1,h))*(w+1)
                    + max(0,min(y+yMinCurr-1,w))]
                + intData[max(0,bClip)*(w+1)
                    + max(0,min(y+yMinCurr-1,w))] );
            
            xMinDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,min(x+xMinCurr-1,h))*(w+1) 
                    + max(0,rClip)]
                - intData[min(h,tClip)*(w+1)
                    + max(0,rClip)]
                - intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,min(y+yMinCurr-1,w))]
                + intData[min(h,tClip)*(w+1)
                    + max(0,min(y+yMinCurr-1,w))] );
            
            yMaxDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,bClip)*(w+1) 
                    + max(0,min(y+yMaxCurr+1,w))]
                - intData[max(0,bClip)*(w+1)
                    + max(0,rClip)]
                - intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,min(y+yMaxCurr+1,w))]
                + intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,rClip)] );
            
            yMinDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,bClip)*(w+1) 
                    + max(0,min(y+yMinCurr-1,w))]
                - intData[max(0,bClip)*(w+1)
                    + min(w,lClip)]
                - intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,min(y+yMinCurr-1,w))]
                + intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + min(w,lClip)] );
        }
    }
}

} // extern "C"
