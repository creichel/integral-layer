
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <algorithm>
#include <cmath>

#define BLOCK_SIZE 32

using std::max;
using std::min;
using std::floor;
using std::ceil;

__global__ void forwardKernelSingle(
    float *intData, float *outData, int h, int w, int xMinCurr, 
    int xMaxCurr, int yMinCurr, int yMaxCurr, float areaCoeff) {

    int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    if (x < h and y < w) {
        int t = max(0, min(x+xMinCurr, h) );
        int b = max(0, min(x+xMaxCurr, h) );
        int l = max(0, min(y+yMinCurr, w) );
        int r = max(0, min(y+yMaxCurr, w) );

        outData[x*w + y] = areaCoeff *
            ( intData[b*(w+1) + r]
            - intData[t*(w+1) + r]
            - intData[b*(w+1) + l]
            + intData[t*(w+1) + l]);
    }
}

__global__ void forwardKernel(
    float *intData, float *outData, int h, int w, 
    float *xMin, float *xMax, float *yMin, float *yMax, float *areaCoeff) {

    int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;
    // below we also mean: // int nWindow = blockIdx.z;

    if (x < h and y < w) {

        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at indices 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.

        int t = max(0, min(x+(int)ceil (xMin[blockIdx.z]  ), h) );
        int b = max(0, min(x+(int)floor(xMax[blockIdx.z]+1), h) );
        int l = max(0, min(y+(int)ceil (yMin[blockIdx.z]  ), w) );
        int r = max(0, min(y+(int)floor(yMax[blockIdx.z]+1), w) );

        outData[blockIdx.z*w*h + x*w + y] = areaCoeff[blockIdx.z] *
            ( intData[b*(w+1) + r]
            - intData[t*(w+1) + r]
            - intData[b*(w+1) + l]
            + intData[t*(w+1) + l]);
    }
}

extern "C" {

void forwardCudaSingle(
    float *intData, int h, int w, float *outData,
    int xMinCurr, int xMaxCurr, int yMinCurr, int yMaxCurr, float areaCoeff) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((h + dimBlock.x - 1) / dimBlock.x, (w + dimBlock.y - 1) / dimBlock.y);

    forwardKernelSingle <<<dimGrid, dimBlock>>> (intData, outData, h, w, xMinCurr, xMaxCurr, yMinCurr, yMaxCurr, areaCoeff);
}

void forwardCuda(
    float *intData, int h, int w, int nWindows, float *outData,
    float *xMin, float *xMax, float *yMin, float *yMax, float *areaCoeff) {

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1024 / (BLOCK_SIZE * BLOCK_SIZE));
    dim3 dimGrid((h + dimBlock.x - 1) / dimBlock.x, (w + dimBlock.y - 1) / dimBlock.y, nWindows);

    forwardKernel <<<dimGrid, dimBlock>>> (intData, outData, h, w, xMin, xMax, yMin, yMax, areaCoeff);
}

void backwardCudaSingle(
    float *intData, float *gradOutData, int h, int w, float *deltas,
    int xMinCurr, int xMaxCurr, int yMinCurr, int yMaxCurr) {

    float & xMaxDelta = deltas[1];
    float & xMinDelta = deltas[0];
    float & yMaxDelta = deltas[3];
    float & yMinDelta = deltas[2];

    for (int x = 1; x <= h; ++x) {
        for (int y = 1; y <= w; ++y) {
            
            int tClip = max(x+xMinCurr, 0);
            int bClip = min(x+xMaxCurr, h);
            int lClip = max(y+yMinCurr, 0);
            int rClip = min(y+yMaxCurr, w);

            xMaxDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,min(x+xMaxCurr+1,h))*(w+1) + max(0,rClip)]
                - intData[max(0,bClip)*(w+1) + max(0,rClip)]
                - intData[max(0,min(x+xMaxCurr+1,h))*(w+1)
                    + max(0,min(y+yMinCurr-1,w))]
                + intData[max(0,bClip)*(w+1)
                    + max(0,min(y+yMinCurr-1,w))] );
            
            xMinDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,min(x+xMinCurr-1,h))*(w+1) 
                    + max(0,rClip)]
                - intData[min(h,tClip)*(w+1)
                    + max(0,rClip)]
                - intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,min(y+yMinCurr-1,w))]
                + intData[min(h,tClip)*(w+1)
                    + max(0,min(y+yMinCurr-1,w))] );
            
            yMaxDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,bClip)*(w+1) 
                    + max(0,min(y+yMaxCurr+1,w))]
                - intData[max(0,bClip)*(w+1)
                    + max(0,rClip)]
                - intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,min(y+yMaxCurr+1,w))]
                + intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,rClip)] );
            
            yMinDelta += gradOutData[(x-1)*w + (y-1)] *
                ( intData[max(0,bClip)*(w+1) 
                    + max(0,min(y+yMinCurr-1,w))]
                - intData[max(0,bClip)*(w+1)
                    + min(w,lClip)]
                - intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + max(0,min(y+yMinCurr-1,w))]
                + intData[max(0,min(x+xMinCurr-1,h))*(w+1)
                    + min(w,lClip)] );
        }
    }
}

} // extern "C"
